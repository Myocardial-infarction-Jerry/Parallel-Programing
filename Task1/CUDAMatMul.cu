#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matMul(float *A, float *B, float *C, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= size || col >= size)
        return;

    float sum = 0;
    for (int i = 0; i < size; ++i)
        sum += A[row * size + i] * B[i * size + col];

    C[row * size + col] = sum;
}

int main(int argc, char const *argv[]) {
    std::cerr << "Enter the size of the matrix:" << std::endl;
    int size; std::cin >> size;

    float *A, *B, *C;
    A = new float[size * size];
    B = new float[size * size];
    C = new float[size * size];

    for (int i = 0; i < size * size; ++i) {
        A[i] = (float)(rand()) / RAND_MAX;
        B[i] = (float)(rand()) / RAND_MAX;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size * size * sizeof(float));
    hipMalloc(&d_B, size * size * sizeof(float));
    hipMalloc(&d_C, size * size * sizeof(float));
    hipMemcpy(d_A, A, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size * size * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(size / threadsPerBlock.x, size / threadsPerBlock.y);
    matMul << <numBlocks, threadsPerBlock >> > (d_A, d_B, d_C, size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(C, d_C, size * size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    std::cout << "Running time: " << milliseconds / 1000 << " seconds" << std::endl;

    return 0;
}